#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "col2im.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void col2im_gpu_kernel(const int n, const float* data_col,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_im) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        float val = 0;
        int w = index % width + pad;
        int h = (index / width) % height + pad;
        int c = index / (width * height);
        // compute the start and end of the output
        int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
        int w_col_end = min(w / stride + 1, width_col);
        int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
        int h_col_end = min(h / stride + 1, height_col);
        // equivalent implementation
        int offset =
            (c * ksize * ksize + h * ksize + w) * height_col * width_col;
        int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
        int coeff_w_col = (1 - stride * height_col * width_col);
        for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
            for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
                val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
            }
        }
        data_im[index] += val;
    }
}

void col2im_gpu(float *data_col,
        int channels, int height, int width,
        int ksize, int stride, int pad, float *data_im){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height * width;
    col2im_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, data_col, height, width, ksize, pad,
                stride, height_col,
                width_col, data_im);
}

