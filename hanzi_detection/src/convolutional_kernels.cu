#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

__global__ void binarize_kernel(float *x, int n, float *binary)
{
    int i = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (i >= n) return;
    binary[i] = (x[i] >= 0) ? 1 : -1;
}

void binarize_gpu(float *x, int n, float *binary)
{
    binarize_kernel<<<cuda_gridsize(n), BLOCK>>>(x, n, binary);
    check_error(hipPeekAtLastError());
}

__global__ void binarize_input_kernel(float *input, int n, int size, float *binary)
{
    int s = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (s >= size) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < n; ++i){
        mean += fabsf(input[i*size + s]);
    }
    mean = mean / n;
    for(i = 0; i < n; ++i){
        binary[i*size + s] = (input[i*size + s] > 0) ? mean : -mean;
    }
}

void binarize_input_gpu(float *input, int n, int size, float *binary)
{
    binarize_input_kernel<<<cuda_gridsize(size), BLOCK>>>(input, n, size, binary);
    check_error(hipPeekAtLastError());
}


__global__ void binarize_weights_kernel(float *weights, int n, int size, float *binary)
{
    int f = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (f >= n) return;
    int i = 0;
    float mean = 0;
    for(i = 0; i < size; ++i){
        mean += fabsf(weights[f*size + i]);
    }
    mean = mean / size;
    for(i = 0; i < size; ++i){
        binary[f*size + i] = (weights[f*size + i] > 0) ? mean : -mean;
        //binary[f*size + i] = weights[f*size + i];
    }
}

void binarize_weights_gpu(float *weights, int n, int size, float *binary)
{
    binarize_weights_kernel<<<cuda_gridsize(n), BLOCK>>>(weights, n, size, binary);
    check_error(hipPeekAtLastError());
}

void forward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);
    if(l.binary){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
    }

    if(l.xnor){
        binarize_weights_gpu(l.weights_gpu, l.n, l.c/l.groups*l.size*l.size, l.binary_weights_gpu);
        swap_binary(&l);
        binarize_gpu(net.input_gpu, l.c*l.h*l.w*l.batch, l.binary_input_gpu);
        net.input_gpu = l.binary_input_gpu;
    }

#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionForward(cudnn_handle(),
                &one,
                l.srcTensorDesc,
                net.input_gpu,
                l.weightDesc,
                l.weights_gpu,
                l.convDesc,
                l.fw_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dstTensorDesc,
                l.output_gpu);

#else
    int i, j;
    int m = l.n/l.groups;
    int k = l.size*l.size*l.c/l.groups;
    int n = l.out_w*l.out_h;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.weights_gpu + j*l.nweights/l.groups;
            float *b = net.workspace;
            float *c = l.output_gpu + (i*l.groups + j)*n*m;
            float *im = net.input_gpu + (i*l.groups + j)*l.c/l.groups*l.h*l.w;

            if (l.size == 1){
                b = im;
            } else {
                im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            }
            gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
#endif

    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    activate_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation);
    //if(l.dot > 0) dot_error_gpu(l);
    if(l.binary || l.xnor) swap_binary(&l);
}

__global__ void smooth_kernel(float *x, int n, int w, int h, int c, int size, float rate, float *delta)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if(id >= n) return;

    int j = id % w;
    id /= w;
    int i = id % h;
    id /= h;
    int k = id % c;
    id /= c;
    int b = id;

    int w_offset = -(size/2.f);
    int h_offset = -(size/2.f);

    int out_index = j + w*(i + h*(k + c*b));
    int l, m;
    for(l = 0; l < size; ++l){
        for(m = 0; m < size; ++m){
            int cur_h = h_offset + i + l;
            int cur_w = w_offset + j + m;
            int index = cur_w + w*(cur_h + h*(k + b*c));
            int valid = (cur_h >= 0 && cur_h < h &&
                    cur_w >= 0 && cur_w < w);
            delta[out_index] += valid ? rate*(x[index] - x[out_index]) : 0;
        }
    }
}

extern "C" void smooth_layer(layer l, int size, float rate)
{
    int h = l.out_h;
    int w = l.out_w;
    int c = l.out_c;

    size_t n = h*w*c*l.batch;

    smooth_kernel<<<cuda_gridsize(n), BLOCK>>>(l.output_gpu, n, l.w, l.h, l.c, size, rate, l.delta_gpu);
    check_error(hipPeekAtLastError());
}

void backward_convolutional_layer_gpu(convolutional_layer l, network net)
{
    if(l.smooth){
        smooth_layer(l, 5, l.smooth);
    }
    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);


    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    float *original_input = net.input_gpu;

    if(l.xnor) net.input_gpu = l.binary_input_gpu;
#ifdef CUDNN
    float one = 1;
    hipdnnConvolutionBackwardFilter(cudnn_handle(),
            &one,
            l.srcTensorDesc,
            net.input_gpu,
            l.ddstTensorDesc,
            l.delta_gpu,
            l.convDesc,
            l.bf_algo,
            net.workspace,
            l.workspace_size,
            &one,
            l.dweightDesc,
            l.weight_updates_gpu);

    if(net.delta_gpu){
        if(l.binary || l.xnor) swap_binary(&l);
        hipdnnConvolutionBackwardData(cudnn_handle(),
                &one,
                l.weightDesc,
                l.weights_gpu,
                l.ddstTensorDesc,
                l.delta_gpu,
                l.convDesc,
                l.bd_algo,
                net.workspace,
                l.workspace_size,
                &one,
                l.dsrcTensorDesc,
                net.delta_gpu);
        if(l.binary || l.xnor) swap_binary(&l);
        if(l.xnor) gradient_array_gpu(original_input, l.batch*l.c*l.h*l.w, HARDTAN, net.delta_gpu);
    }

#else
    int m = l.n/l.groups;
    int n = l.size*l.size*l.c/l.groups;
    int k = l.out_w*l.out_h;

    int i, j;
    for(i = 0; i < l.batch; ++i){
        for(j = 0; j < l.groups; ++j){
            float *a = l.delta_gpu + (i*l.groups + j)*m*k;
            float *b = net.workspace;
            float *c = l.weight_updates_gpu + j*l.nweights/l.groups;

            float *im  = net.input_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;
            float *imd = net.delta_gpu+(i*l.groups + j)*l.c/l.groups*l.h*l.w;

            im2col_gpu(im, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, b);
            gemm_gpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

            if (net.delta_gpu) {
                if (l.binary || l.xnor) swap_binary(&l);
                a = l.weights_gpu + j*l.nweights/l.groups;
                b = l.delta_gpu + (i*l.groups + j)*m*k;
                c = net.workspace;
                if (l.size == 1) {
                    c = imd;
                }

                gemm_gpu(1,0,n,k,m,1,a,n,b,k,0,c,k);

                if (l.size != 1) {
                    col2im_gpu(net.workspace, l.c/l.groups, l.h, l.w, l.size, l.stride, l.pad, imd);
                }
                if(l.binary || l.xnor) {
                    swap_binary(&l);
                }
            }
            if(l.xnor) gradient_array_gpu(original_input + i*l.c*l.h*l.w, l.c*l.h*l.w, HARDTAN, net.delta_gpu + i*l.c*l.h*l.w);
        }
    }
#endif
}

void pull_convolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.nweights);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void push_convolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.nweights);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.nweights);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_convolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        axpy_gpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
    if(l.clip){
        constrain_gpu(l.nweights, l.clip, l.weights_gpu, 1);
    }
}


