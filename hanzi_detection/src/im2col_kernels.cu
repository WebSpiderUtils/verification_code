#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "im2col.h"
#include "hip/hip_runtime.h"
}

// src: https://github.com/BVLC/caffe/blob/master/src/caffe/util/im2col.cu
// You may also want to read: https://github.com/BVLC/caffe/blob/master/LICENSE

__global__ void im2col_gpu_kernel(const int n, const float* data_im,
        const int height, const int width, const int ksize,
        const int pad,
        const int stride,
        const int height_col, const int width_col,
        float *data_col) {
    int index = blockIdx.x*blockDim.x+threadIdx.x;
    for(; index < n; index += blockDim.x*gridDim.x){
        int w_out = index % width_col;
        int h_index = index / width_col;
        int h_out = h_index % height_col;
        int channel_in = h_index / height_col;
        int channel_out = channel_in * ksize * ksize;
        int h_in = h_out * stride - pad;
        int w_in = w_out * stride - pad;
        float* data_col_ptr = data_col;
        data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
        const float* data_im_ptr = data_im;
        data_im_ptr += (channel_in * height + h_in) * width + w_in;
        for (int i = 0; i < ksize; ++i) {
            for (int j = 0; j < ksize; ++j) {
                int h = h_in + i;
                int w = w_in + j;

                *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
                    data_im_ptr[i * width + j] : 0;

                //*data_col_ptr = data_im_ptr[ii * width + jj];

                data_col_ptr += height_col * width_col;
            }
        }
    }
}

void im2col_gpu(float *im,
         int channels, int height, int width,
         int ksize, int stride, int pad, float *data_col){
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int height_col = (height + 2 * pad - ksize) / stride + 1;
    int width_col = (width + 2 * pad - ksize) / stride + 1;
    int num_kernels = channels * height_col * width_col;
    im2col_gpu_kernel<<<(num_kernels+BLOCK-1)/BLOCK,
        BLOCK>>>(
                num_kernels, im, height, width, ksize, pad,
                stride, height_col,
                width_col, data_col);
}
