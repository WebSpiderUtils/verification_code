#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"

extern "C" {
#include "convolutional_layer.h"
#include "deconvolutional_layer.h"
#include "batchnorm_layer.h"
#include "gemm.h"
#include "blas.h"
#include "im2col.h"
#include "col2im.h"
#include "utils.h"
#include "hip/hip_runtime.h"
}

extern "C" void forward_deconvolutional_layer_gpu(layer l, network net)
{
    int i;

    int m = l.size*l.size*l.n;
    int n = l.h*l.w;
    int k = l.c;

    fill_gpu(l.outputs*l.batch, 0, l.output_gpu, 1);

    for(i = 0; i < l.batch; ++i){
        float *a = l.weights_gpu;
        float *b = net.input_gpu + i*l.c*l.h*l.w;
        float *c = net.workspace;

        gemm_gpu(1,0,m,n,k,1,a,m,b,n,0,c,n);

        col2im_gpu(net.workspace, l.out_c, l.out_h, l.out_w, l.size, l.stride, l.pad, l.output_gpu+i*l.outputs);
    }
    if (l.batch_normalize) {
        forward_batchnorm_layer_gpu(l, net);
    } else {
        add_bias_gpu(l.output_gpu, l.biases_gpu, l.batch, l.n, l.out_w*l.out_h);
    }
    activate_array_gpu(l.output_gpu, l.batch*l.n*l.out_w*l.out_h, l.activation);
}

extern "C" void backward_deconvolutional_layer_gpu(layer l, network net)
{
    int i;

    //constrain_gpu(l.outputs*l.batch, 1, l.delta_gpu, 1);
    gradient_array_gpu(l.output_gpu, l.outputs*l.batch, l.activation, l.delta_gpu);

    if(l.batch_normalize){
        backward_batchnorm_layer_gpu(l, net);
    } else {
        backward_bias_gpu(l.bias_updates_gpu, l.delta_gpu, l.batch, l.n, l.out_w*l.out_h);
    }

    //if(net.delta_gpu) memset(net.delta_gpu, 0, l.batch*l.h*l.w*l.c*sizeof(float));

    for(i = 0; i < l.batch; ++i){
        int m = l.c;
        int n = l.size*l.size*l.n;
        int k = l.h*l.w;

        float *a = net.input_gpu + i*m*k;
        float *b = net.workspace;
        float *c = l.weight_updates_gpu;

        im2col_gpu(l.delta_gpu + i*l.outputs, l.out_c, l.out_h, l.out_w, 
                l.size, l.stride, l.pad, b);
        gemm_gpu(0,1,m,n,k,1,a,k,b,k,1,c,n);

        if(net.delta_gpu){
            int m = l.c;
            int n = l.h*l.w;
            int k = l.size*l.size*l.n;

            float *a = l.weights_gpu;
            float *b = net.workspace;
            float *c = net.delta_gpu + i*n*m;

            gemm_gpu(0,0,m,n,k,1,a,k,b,n,1,c,n);
        }
    }
}

extern "C" void pull_deconvolutional_layer(layer l)
{
    cuda_pull_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.biases_gpu, l.biases, l.n);
    cuda_pull_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_pull_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_pull_array(l.scales_gpu, l.scales, l.n);
        cuda_pull_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_pull_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

extern "C" void push_deconvolutional_layer(layer l)
{
    cuda_push_array(l.weights_gpu, l.weights, l.c*l.n*l.size*l.size);
    cuda_push_array(l.biases_gpu, l.biases, l.n);
    cuda_push_array(l.weight_updates_gpu, l.weight_updates, l.c*l.n*l.size*l.size);
    cuda_push_array(l.bias_updates_gpu, l.bias_updates, l.n);
    if (l.batch_normalize){
        cuda_push_array(l.scales_gpu, l.scales, l.n);
        cuda_push_array(l.rolling_mean_gpu, l.rolling_mean, l.n);
        cuda_push_array(l.rolling_variance_gpu, l.rolling_variance, l.n);
    }
}

void update_deconvolutional_layer_gpu(layer l, update_args a)
{
    float learning_rate = a.learning_rate*l.learning_rate_scale;
    float momentum = a.momentum;
    float decay = a.decay;
    int batch = a.batch;

    if(a.adam){
        adam_update_gpu(l.weights_gpu, l.weight_updates_gpu, l.m_gpu, l.v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.nweights, batch, a.t);
        adam_update_gpu(l.biases_gpu, l.bias_updates_gpu, l.bias_m_gpu, l.bias_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        if(l.scales_gpu){
            adam_update_gpu(l.scales_gpu, l.scale_updates_gpu, l.scale_m_gpu, l.scale_v_gpu, a.B1, a.B2, a.eps, decay, learning_rate, l.n, batch, a.t);
        }
    }else{
        axpy_gpu(l.nweights, -decay*batch, l.weights_gpu, 1, l.weight_updates_gpu, 1);
        axpy_gpu(l.nweights, learning_rate/batch, l.weight_updates_gpu, 1, l.weights_gpu, 1);
        scal_gpu(l.nweights, momentum, l.weight_updates_gpu, 1);

        axpy_gpu(l.n, learning_rate/batch, l.bias_updates_gpu, 1, l.biases_gpu, 1);
        scal_gpu(l.n, momentum, l.bias_updates_gpu, 1);

        if(l.scales_gpu){
            axpy_gpu(l.n, learning_rate/batch, l.scale_updates_gpu, 1, l.scales_gpu, 1);
            scal_gpu(l.n, momentum, l.scale_updates_gpu, 1);
        }
    }
}

